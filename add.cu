
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <optional>
#include <limits>
#include <iomanip>


// Demonstration kernels (non-deterministic):
// These kernels use different reduction orders/parallelism to show
// floating-point non-determinism.
// One thread per row, reducing forwards
template <int M, int N>
__global__
void reduce_rows_single_for(float *X, float *out) {
  int row = threadIdx.x;
  if (row >= M) return;

  float res = 0.0f;
  for (int col = 0; col < N; ++col) {
    res += X[row * N + col];
  }
  out[row] = res;
}

// One thread per row, reducing backwards
template <int M, int N>
__global__
void reduce_rows_single_back(float *X, float *out) {
  int row = threadIdx.x;
  if (row >= M) return;

  float res = 0.0f;
  for (int col = N - 1; col >= 0; --col) {
    res += X[row * N + col];
  }
  out[row] = res;
}

// 16 threads per row (non-deterministic)
// Each thread reduces a contiguous chunk of N/16 elements
template <int M, int N>
__global__
void reduce_rows_par_16(float *X, float *out) {
  __shared__ float temp[16];
  int row = blockIdx.x;
  int tid = threadIdx.x;
  
  float sum = 0.0f;
  for (int i = tid * (N/16); i < (tid + 1) * (N/16); ++i) {
    sum += X[row * N + i];
  }
  temp[tid] = sum;
  __syncthreads();

  // Reduction in shared memory
  if (tid == 0) {
    float total = 0.0f;
    for (int i = 0; i < 16; ++i) {
      total += temp[i];
    }
    out[row] = total;
  }
}

// 64 threads per row (non-deterministic)
// Each thread reduces a contiguous chunk of N/64 elements
template <int M, int N>
__global__
void reduce_rows_par_64(float *X, float *out) {
  __shared__ float temp[64];
  int row = blockIdx.x;
  int tid = threadIdx.x;
  
  float sum = 0.0f;
  for (int i = tid * (N/64); i < (tid + 1) * (N/64); ++i) {
    sum += X[row * N + i];
  }
  temp[tid] = sum;
  __syncthreads();

  // Reduction in shared memory
  if (tid == 0) {
    float total = 0.0f;
    for (int i = 0; i < 64; ++i) {
      total += temp[i];
    }
    out[row] = total;
  }
}

// 256 threads per row (non-deterministic)
// Each thread reduces a contiguous chunk of N/256 elements
// 
// The reduction order between the chunks is a binary-tree
// While this isn't maximally efficient (loads aren't coalesced),
// it lets coarser kernels easily do some of the work of combining
// chunk results within each thread.
template <int M, int N>
__global__
void reduce_rows_par_256(float *X, float *out) {
  __shared__ float temp[256];
  int row = blockIdx.x;
  int tid = threadIdx.x;
  
  float sum = 0.0f;
  for (int i = tid * (N/256); i < (tid + 1) * (N/256); ++i) {
    sum += X[row * N + i];
  }
  temp[tid] = sum;
  __syncthreads();

  // Binary tree reduction in shared memory
  for (int stride = 1; stride < 256; stride *= 2) {
    if (tid % (stride * 2) == 0) {
      temp[tid] += temp[tid + stride];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[row] = temp[0];
  }
}

// Deterministic kernels (match reduce_rows_par_256):
// These kernels compute chunked sums in the same manner as
// reduce_rows_par_256, and then combines chunk same in the
// same binary tree, so their results match exactly


// 64 threads per row (deterministic)
// Each thread reduces multiple chunks of size N/256
template <int M, int N>
__global__
void reduce_rows_par_64_determ(float *X, float *out) {
  __shared__ float temp[64]; // multi-chunk results across threads
  int row = blockIdx.x;
  int tid = threadIdx.x;
  int chunks = 256;
  int thread_chunks = 4; // = chunks / 64, chunks summed by this thread
  int chunk_size = N/chunks;

  float chunk_sums[4] = {0.0f};
  int base = tid * (N/64);
  for (int cidx = 0; cidx < thread_chunks; ++cidx) {
    for (int i = base + cidx * chunk_size; i < base + (cidx + 1) * chunk_size; ++i) {
      chunk_sums[cidx] += X[row * N + i];
    }
  }
  temp[tid] = (chunk_sums[0] + chunk_sums[1]) + (chunk_sums[2] + chunk_sums[3]);

  __syncthreads();

  // Binary tree reduction in shared memory
  for (int stride = 1; stride < 64; stride *= 2) {
    if (tid % (stride * 2) == 0) {
      temp[tid] += temp[tid + stride];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[row] = temp[0];
  }
}


// 16 threads per row (deterministic)
// Each thread reduces multiple chunks of size N/256
template <int M, int N>
__global__
void reduce_rows_par_16_determ(float *X, float *out) {
  __shared__ float temp[16]; // multi-chunk results across threads
  int row = blockIdx.x;
  int tid = threadIdx.x;
  int chunks = 256;
  int thread_chunks = 16; // = chunks / 16, chunks summed by this thread
  int chunk_size = N/chunks;

  float chunk_sums[16] = {0.0f};
  int base = tid * (N/16);
  for (int cidx = 0; cidx < thread_chunks; ++cidx) {
    for (int i = base + cidx * chunk_size; i < base + (cidx + 1) * chunk_size; ++i) {
      chunk_sums[cidx] += X[row * N + i];
    }
  }
  // Binary reduction over the chunk sums
  temp[tid] = ((chunk_sums[0] + chunk_sums[1]) + (chunk_sums[2] + chunk_sums[3])) +
        ((chunk_sums[4] + chunk_sums[5]) + (chunk_sums[6] + chunk_sums[7])) +
        ((chunk_sums[8] + chunk_sums[9]) + (chunk_sums[10] + chunk_sums[11])) +
        ((chunk_sums[12] + chunk_sums[13]) + (chunk_sums[14] + chunk_sums[15]));
  __syncthreads();

  // Binary tree reduction in shared memory
  // TODO: Warp shuffle this instead
  for (int stride = 1; stride < 16; stride *= 2) {
    if (tid % (stride * 2) == 0) {
      temp[tid] += temp[tid + stride];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[row] = temp[0];
  }
}


// 1 thread per row (deterministic)
// Reduce a row by summing chunks of size N/256 in the same order
// used by reduce_rows_par_256
template <int M, int N>
__global__
void reduce_rows_single_determ(float *X, float *out) {
  int row = threadIdx.x;
  if (row >= M) return;

  int chunks = 256;
  int chunk_size = N/chunks;
  // need storage for all chunk sums
  float chunk_sums[256] = {0.0f};

  // obeying the order of reduce_rows_par_256
  for (int cidx = 0; cidx < chunks; ++cidx) {
    for (int i = cidx * chunk_size; i < (cidx + 1) * chunk_size; ++i) {
      chunk_sums[cidx] += X[row * N + i];
    }
  }

  // Binary tree reduction over chunk_sums
  for (int stride = 1; stride < chunks; stride *= 2) {
    for (int i = 0; i < chunks; i += stride * 2) {
      chunk_sums[i] += chunk_sums[i + stride];
    }
  }
  out[row] = chunk_sums[0];
}

std::mt19937 make_rng(std::optional<unsigned int> seed = std::nullopt) {
  if (seed) return std::mt19937(*seed);
  else {
    std::random_device rd;
    return std::mt19937(rd());
  }
}

// Initialize matrix X (M x N) with samples from a uniform distribution
void init_matrix_std_normal(float *X, int M, int N, std::optional<unsigned int> seed = std::nullopt) {
  std::mt19937 gen = make_rng(seed);
  // Use a much smaller symmetric range to avoid overflow when summing many
  // elements, while still producing large variance to highlight
  // non-associativity. Adjust as needed.
  float lo = -1e3f;
  float hi = 1e3f;
  std::uniform_real_distribution<float> dist(lo, hi);

  int total = M * N;
  for (int i = 0; i < total; ++i) {
    X[i] = dist(gen);
  }
}

// Helper to print M results with a label
void print_results(const char *label, float *out, int M) {
  std::cout << "\n" << label << ":" << std::endl;
  for (int i = 0; i < M; ++i) {
    std::cout << out[i];
    if (i + 1 < M) std::cout << ", ";
  }
  std::cout << std::endl;
}

int main(int argc, char* argv[]) {
  std::optional<unsigned int> seed;

  if (argc > 1) {
    try {
      seed = std::stoul(argv[1]);
    } catch (const std::exception& e) {
      std::cerr << "Invalid seed: " << argv[1] << '\n';
      return 1;
    }
  }
  constexpr int M = 4;
  constexpr int N = 16384;

  // allocate matrix X and initialize on host
  float *X = nullptr;
  hipMallocManaged(&X, M * N * sizeof(float));
  init_matrix_std_normal(X, M, N, seed);

  hipEvent_t start, stop;
  float ms;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // single reusable output buffer for all kernels
  float *out = nullptr;
  hipMallocManaged(&out, M * sizeof(float));

  // print with higher precision to highlight small differences
  std::cout << std::setprecision(6) << std::scientific;

  auto reset_out = [&](void) {
    for (int i = 0; i < M; ++i) out[i] = 0.0f;
  };

  // SINGLE thread, FORWARD
  reset_out();
  hipEventRecord(start);
  reduce_rows_single_for<M, N><<<1, M>>>(X, out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  print_results("SINGLE thread, FORWARD", out, M);
  std::cout << "Kernel execution time: " << ms << "ms" << std::endl;

  // SINGLE thread, BACKWARD
  reset_out();
  hipEventRecord(start);
  reduce_rows_single_back<M, N><<<1, M>>>(X, out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  print_results("SINGLE thread, BACKWARD", out, M);
  std::cout << "Kernel execution time: " << ms << "ms" << std::endl;

  // PAR 16 threads/row
  reset_out();
  hipEventRecord(start);
  reduce_rows_par_16<M, N><<<M, 16>>>(X, out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  print_results("PAR 16 threads/row", out, M);
  std::cout << "Kernel execution time: " << ms << "ms" << std::endl;

  // PAR 64 threads/row
  reset_out();
  hipEventRecord(start);
  reduce_rows_par_64<M, N><<<M, 64>>>(X, out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  print_results("PAR 64 threads/row", out, M);
  std::cout << "Kernel execution time: " << ms << "ms" << std::endl;

  // PAR 256 threads/row
  reset_out();
  hipEventRecord(start);
  reduce_rows_par_256<M, N><<<M, 256>>>(X, out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  print_results("PAR 256 threads/row", out, M);
  std::cout << "Kernel execution time: " << ms << "ms" << std::endl;

  // SINGLE thread, DETERM
  reset_out();
  hipEventRecord(start);
  reduce_rows_single_determ<M, N><<<1, M>>>(X, out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  print_results("SINGLE thread, DETERM", out, M);
  std::cout << "Kernel execution time: " << ms << "ms" << std::endl;

  // PAR 16 threads/row, DETERM
  reset_out();
  hipEventRecord(start);
  reduce_rows_par_16_determ<M, N><<<M, 16>>>(X, out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  print_results("PAR 16 threads/row, DETERM", out, M);
  std::cout << "Kernel execution time: " << ms << "ms" << std::endl;

  // PAR 64 threads/row, DETERM
  reset_out();
  hipEventRecord(start);
  reduce_rows_par_64_determ<M, N><<<M, 64>>>(X, out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  print_results("PAR 64 threads/row, DETERM", out, M);
  std::cout << "Kernel execution time: " << ms << "ms" << std::endl;

  // destroy events
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(out);
  hipFree(X);
  return 0;
}